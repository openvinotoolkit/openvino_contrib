#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include "acos.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

namespace cumath = CUDA::math;

template <typename T>
struct AcosOpImpl {
    __device__ static inline T op(T x) {
        return cumath::acos(x);
    }
};

Acos::Acos(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : impl_{element_type, max_threads_per_block, num_elements} {}

void Acos::operator()(hipStream_t stream, const void* in0, void* out) const {
    impl_(stream, in0, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
