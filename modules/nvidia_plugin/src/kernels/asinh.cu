#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include "asinh.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

namespace cumath = CUDA::math;

template <typename T>
struct AsinhOpImpl {
    __device__ static inline T op(T x) {
        return cumath::asinh(x);
    }
};

Asinh::Asinh(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : impl_{element_type, max_threads_per_block, num_elements} {}

void Asinh::operator()(hipStream_t stream, const void* in0, void* out) const {
    impl_(stream, in0, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
