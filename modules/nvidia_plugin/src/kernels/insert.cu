#include "hip/hip_runtime.h"
// Copyright (C) 2018-2024 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <fmt/format.h>

#include <cuda/float16.hpp>

#include "details/error.hpp"
#include "details/tensor_helpers.hpp"
#include "details/type_validator.hpp"
#include "insert.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
static __global__ void insert_part(
    const Insert::Props* props, const size_t start, const size_t size, const T* x, T* y) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        const size_t old_rank = rank(props->old_shape);
        const size_t new_rank = rank(props->new_shape);
        assert(old_rank == new_rank);
        Shape<size_t, 5> originalIndexes{};
        shape_indices(props->old_shape, i, originalIndexes);
        Shape<size_t, 5> joinIndexes{};
        memcpy(joinIndexes, originalIndexes, sizeof(originalIndexes));
        joinIndexes[props->axe] = start + joinIndexes[props->axe];
        const size_t flatInputAddress = flat_address_by_shape(props->new_shape, joinIndexes);
        y[flatInputAddress] = x[i];
    }
}

Insert::Insert(const Type_t element_type, const Props& props, const size_t max_threads_per_block)
    : element_type_{element_type}, props_{props}, size_{shape_size(props.old_shape)} {
    TypeValidator<AllElementTypesSwitch>::check(element_type_);
    std::tie(num_blocks_, threads_per_block_) = calculateElementwiseGrid(size_, max_threads_per_block);
}

void Insert::operator()(const hipStream_t stream, const void* src, void* dst, const size_t start) const {
    switch (element_type_) {
        case Type_t::boolean:
            return call<bool>(stream, src, dst, start);
#ifdef CUDA_HAS_BF16_TYPE
        case Type_t::bf16:
            return call<__hip_bfloat16>(stream, src, dst, start);
#endif
        case Type_t::f16:
            return call<__half>(stream, src, dst, start);
        case Type_t::f32:
            return call<float>(stream, src, dst, start);
        case Type_t::f64:
            return call<double>(stream, src, dst, start);
        case Type_t::i8:
            return call<int8_t>(stream, src, dst, start);
        case Type_t::i16:
            return call<int16_t>(stream, src, dst, start);
        case Type_t::i32:
            return call<int32_t>(stream, src, dst, start);
        case Type_t::i64:
            return call<int64_t>(stream, src, dst, start);
        case Type_t::u8:
            return call<uint8_t>(stream, src, dst, start);
        case Type_t::u16:
            return call<uint16_t>(stream, src, dst, start);
        case Type_t::u32:
            return call<uint32_t>(stream, src, dst, start);
        case Type_t::u64:
            return call<uint64_t>(stream, src, dst, start);
        default:
            throw_ov_exception(fmt::format("Input element type = {} is not supported by Insert operation !!",
                                         static_cast<Type_t>(element_type_)));
    }
}

void* Insert::getKernel() const {
    switch (element_type_) {
        case Type_t::boolean:
            return reinterpret_cast<void*>(&insert_part<bool>);
#ifdef CUDA_HAS_BF16_TYPE
        case Type_t::bf16:
            return reinterpret_cast<void*>(&insert_part<__hip_bfloat16>);
#endif
        case Type_t::f16:
            return reinterpret_cast<void*>(&insert_part<__half>);
        case Type_t::f32:
            return reinterpret_cast<void*>(&insert_part<float>);
        case Type_t::f64:
            return reinterpret_cast<void*>(&insert_part<double>);
        case Type_t::i8:
            return reinterpret_cast<void*>(&insert_part<int8_t>);
        case Type_t::i16:
            return reinterpret_cast<void*>(&insert_part<int16_t>);
        case Type_t::i32:
            return reinterpret_cast<void*>(&insert_part<int32_t>);
        case Type_t::i64:
            return reinterpret_cast<void*>(&insert_part<int64_t>);
        case Type_t::u8:
            return reinterpret_cast<void*>(&insert_part<uint8_t>);
        case Type_t::u16:
            return reinterpret_cast<void*>(&insert_part<uint16_t>);
        case Type_t::u32:
            return reinterpret_cast<void*>(&insert_part<uint32_t>);
        case Type_t::u64:
            return reinterpret_cast<void*>(&insert_part<uint64_t>);
        default:
            throw_ov_exception(fmt::format("Input element type = {} is not supported by Insert operation !!",
                                           static_cast<Type_t>(element_type_)));
    }
}

template <typename T>
void Insert::call(const hipStream_t stream, const void* src, void* dst, const size_t start) const {
    assertThrow(props_ptr_, "props_ptr_ == nullptr");
    insert_part<T><<<num_blocks_, threads_per_block_, 0, stream>>>(
        static_cast<const Props*>(props_ptr_), start, size_, static_cast<const T*>(src), static_cast<T*>(dst));
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
